#include "hip/hip_runtime.h"
#include "Common.cuh"
#include "Potf2Kernel.cuh"


inline int get_lds_size() {
  int const default_lds_size = 64 * 1024;

  int lds_size = 0;
  int deviceId = 0;
  auto istat_device = hipGetDevice(&deviceId);
  if (istat_device != hipSuccess) {
    return (default_lds_size);
  };
  auto const attr = hipDeviceAttributeMaxSharedMemoryPerBlock;
  auto istat_attr = hipDeviceGetAttribute(&lds_size, attr, deviceId);
  if (istat_attr != hipSuccess) {
    return (default_lds_size);
  };

  return (lds_size);
}

template <typename U>
__global__ void chk_positive(int *iinfo, int *info, int j, int batch_count) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < batch_count && info[id] == 0 && iinfo[id] > 0)
    info[id] = iinfo[id] + j;
}

template <bool BATCHED, bool STRIDED, typename T>
void rocsolver_potrf_getMemorySize(const int n,
                                   const hipblasFillMode_t uplo,
                                   const int batch_count,
                                   size_t* size_scalars,
                                   size_t* size_work1,
                                   size_t* size_work2,
                                   size_t* size_work3,
                                   size_t* size_work4,
                                   size_t* size_pivots,
                                   size_t* size_iinfo,
                                   bool* optim_mem)
{
    // if quick return no need of workspace
    if(n == 0 || batch_count == 0)
    {
        *size_scalars = 0;
        *size_work1 = 0;
        *size_work2 = 0;
        *size_work3 = 0;
        *size_work4 = 0;
        *size_pivots = 0;
        *size_iinfo = 0;
        *optim_mem = true;
        return;
    }

    int nb = POTRF_BLOCKSIZE(T);
    if(n <= POTRF_POTF2_SWITCHSIZE(T))
    {
        // requirements for calling a single POTF2
        rocsolver_potf2_getMemorySize<T>(n, batch_count, size_scalars, size_work1, size_pivots);
        *size_work2 = 0;
        *size_work3 = 0;
        *size_work4 = 0;
        *size_iinfo = 0;
        *optim_mem = true;
    }
    else
    {
        int jb = nb;
        size_t s1, s2;

        // size to store info about positiveness of each subblock
        *size_iinfo = sizeof(int) * batch_count;

        // requirements for calling POTF2 for the subblocks
        rocsolver_potf2_getMemorySize<T>(jb, batch_count, size_scalars, &s1, size_pivots);

        // extra requirements for calling TRSM
        if(uplo == HIPBLAS_FILL_MODE_UPPER)
        {
            rocsolver_trsm_mem<BATCHED, STRIDED, T>(
                rocblas_side_left, rocblas_operation_conjugate_transpose, jb, n - jb, batch_count,
                &s2, size_work2, size_work3, size_work4, optim_mem);
        }
        else
        {
            rocsolver_trsm_mem<BATCHED, STRIDED, T>(
                rocblas_side_right, rocblas_operation_conjugate_transpose, n - jb, jb, batch_count,
                &s2, size_work2, size_work3, size_work4, optim_mem);
        }

        *size_work1 = std::max(s1, s2);
    }
}

template <bool BATCHED, bool STRIDED, typename T, typename S, typename U>
rocblas_status rocsolver_potrf_template(rocblas_handle handle,
                                        const rocblas_fill uplo,
                                        const int n,
                                        U A,
                                        const int shiftA,
                                        const int lda,
                                        const rocblas_stride strideA,
                                        int* info,
                                        const int batch_count,
                                        T* scalars,
                                        void* work1,
                                        void* work2,
                                        void* work3,
                                        void* work4,
                                        T* pivots,
                                        int* iinfo,
                                        bool optim_mem)
{
    ROCSOLVER_ENTER("potrf", "uplo:", uplo, "n:", n, "shiftA:", shiftA, "lda:", lda,
                    "bc:", batch_count);

    // quick return
    if(batch_count == 0)
        return rocblas_status_success;

    hipStream_t stream;
    rocblas_get_stream(handle, &stream);

    int blocksReset = (batch_count - 1) / BS1 + 1;
    dim3 gridReset(blocksReset, 1, 1);
    dim3 threads(BS1, 1, 1);

    // info=0 (starting with a positive definite matrix)
    ROCSOLVER_LAUNCH_KERNEL(reset_info, gridReset, threads, 0, stream, info, batch_count, 0);

    // quick return
    if(n == 0)
        return rocblas_status_success;

    // everything must be executed with scalars on the host
    rocblas_pointer_mode old_mode;
    rocblas_get_pointer_mode(handle, &old_mode);
    rocblas_set_pointer_mode(handle, rocblas_pointer_mode_host);

    // if the matrix is small, use the unblocked (BLAS-levelII) variant of the
    // algorithm
    int nb = POTRF_BLOCKSIZE(T);
    if(n <= POTRF_POTF2_SWITCHSIZE(T))
        return rocsolver_potf2_template<T>(handle, uplo, n, A, shiftA, lda, strideA, info,
                                           batch_count, scalars, (T*)work1, pivots);

    // constants for rocblas functions calls
    T t_one = 1;
    S s_one = 1;
    S s_minone = -1;

    int jb, j = 0;

    // (TODO: When the matrix is detected to be non positive definite, we need to
    //  prevent TRSM and HERK to modify further the input matrix; ideally with no
    //  synchronizations.)

    if(uplo == rocblas_fill_upper)
    {
        // Compute the Cholesky factorization A = U'*U.
        while(j < n - POTRF_POTF2_SWITCHSIZE(T))
        {
            // Factor diagonal and subdiagonal blocks
            jb = std::min(n - j, nb); // number of columns in the block
            ROCSOLVER_LAUNCH_KERNEL(reset_info, gridReset, threads, 0, stream, iinfo, batch_count, 0);
            rocsolver_potf2_template<T>(handle, uplo, jb, A, shiftA + idx2D(j, j, lda), lda,
                                        strideA, iinfo, batch_count, scalars, (T*)work1, pivots);

            // test for non-positive-definiteness.
            ROCSOLVER_LAUNCH_KERNEL(chk_positive<U>, gridReset, threads, 0, stream, iinfo, info, j,
                                    batch_count);

            if(j + jb < n)
            {
                // update trailing submatrix
                rocsolver_trsm_upper<BATCHED, STRIDED, T>(
                    handle, rocblas_side_left, rocblas_operation_conjugate_transpose,
                    rocblas_diagonal_non_unit, jb, (n - j - jb), A, shiftA + idx2D(j, j, lda), lda,
                    strideA, A, shiftA + idx2D(j, j + jb, lda), lda, strideA, batch_count,
                    optim_mem, work1, work2, work3, work4);

                rocblasCall_syrk_herk<BATCHED, T>(
                    handle, uplo, rocblas_operation_conjugate_transpose, n - j - jb, jb, &s_minone,
                    A, shiftA + idx2D(j, j + jb, lda), lda, strideA, &s_one, A,
                    shiftA + idx2D(j + jb, j + jb, lda), lda, strideA, batch_count);
            }
            j += nb;
        }
    }
    else
    {
        // Compute the Cholesky factorization A = L*L'.
        while(j < n - POTRF_POTF2_SWITCHSIZE(T))
        {
            // Factor diagonal and subdiagonal blocks
            jb = std::min(n - j, nb); // number of columns in the block
            ROCSOLVER_LAUNCH_KERNEL(reset_info, gridReset, threads, 0, stream, iinfo, batch_count, 0);
            rocsolver_potf2_template<T>(handle, uplo, jb, A, shiftA + idx2D(j, j, lda), lda,
                                        strideA, iinfo, batch_count, scalars, (T*)work1, pivots);

            // test for non-positive-definiteness.
            ROCSOLVER_LAUNCH_KERNEL(chk_positive<U>, gridReset, threads, 0, stream, iinfo, info, j,
                                    batch_count);

            if(j + jb < n)
            {
                // update trailing submatrix
                rocsolver_trsm_lower<BATCHED, STRIDED, T>(
                    handle, rocblas_side_right, rocblas_operation_conjugate_transpose,
                    rocblas_diagonal_non_unit, (n - j - jb), jb, A, shiftA + idx2D(j, j, lda), lda,
                    strideA, A, shiftA + idx2D(j + jb, j, lda), lda, strideA, batch_count,
                    optim_mem, work1, work2, work3, work4);

                rocblasCall_syrk_herk<BATCHED, T>(
                    handle, uplo, rocblas_operation_none, n - j - jb, jb, &s_minone, A,
                    shiftA + idx2D(j + jb, j, lda), lda, strideA, &s_one, A,
                    shiftA + idx2D(j + jb, j + jb, lda), lda, strideA, batch_count);
            }
            j += nb;
        }
    }

    // factor last block
    if(j < n)
    {
        rocsolver_potf2_template<T>(handle, uplo, n - j, A, shiftA + idx2D(j, j, lda), lda, strideA,
                                    iinfo, batch_count, scalars, (T*)work1, pivots);
        ROCSOLVER_LAUNCH_KERNEL(chk_positive<U>, gridReset, threads, 0, stream, iinfo, info, j,
                                batch_count);
    }

    rocblas_set_pointer_mode(handle, old_mode);
    return rocblas_status_success;
}
